
#include <hip/hip_runtime.h>
__global__ void copyTileToCanvas(unsigned char* canvas, int canvasWidth, int canvasHeight,
    unsigned char* tile, int tileWidth, int tileHeight,
    int offsetX, int offsetY, int canvasTileWidth, int canvasTileHeight)
{
    // Calculate the global x and y index for the thread
    int canvasX = blockIdx.x * blockDim.x + threadIdx.x;
    int canvasY = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within the bounds of the canvas
    if (canvasX < canvasTileWidth && canvasY < canvasTileHeight) {
        // Translate canvas coordinates into tile coordinates
        float tileX = ((float)canvasX / canvasTileWidth) * tileWidth;
        float tileY = ((float)canvasY / canvasTileHeight) * tileHeight;

        // Find nearest tile pixel for scaling (nearest neighbor scaling)
        int srcX = (int)tileX;
        int srcY = (int)tileY;

        // Ensure the tile indices are within bounds
        if (srcX < tileWidth && srcY < tileHeight) {
            // Calculate the destination index for the canvas
            int canvasIdx = ((canvasY + offsetY) * canvasWidth + (canvasX + offsetX)) * 3;

            // Calculate the source index for the tile
            int tileIdx = (srcY * tileWidth + srcX) * 3;

            // Ensure the canvas indices are within bounds
            if (canvasX + offsetX < canvasWidth && canvasY + offsetY < canvasHeight) {
                // Copy the pixel (RGB components) from tile to canvas
                canvas[canvasIdx] = tile[tileIdx];
                canvas[canvasIdx + 1] = tile[tileIdx + 1];
                canvas[canvasIdx + 2] = tile[tileIdx + 2];
            }
        }
    }
}
